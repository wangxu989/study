#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

int reduction(int a) {
  a += shlf_xor(a, 16);
  a += shlf_xor(a, 8);
  a += shfl_xor(a, 4);
  a += shfl_xor(a, 2);
  a += shfl_xor(a, 1);
  return a;
}
void h_matmul(int *a, int* b,int* target, int height, int width, int width2, int n ) {
  for(int i = 0;i < height;++i) {
    for (int j = 0;j < width2;++j) {
      for (int k = 0;k < width;++k) {
        target[i * width2 + j] += a[i * height + k] * b[width2 * k + j];
      }
    }
  }
}
template<unsigned int per_deal>
void __global__ matmul(int *a, int* b,int* target, int height, int width, width2, int n ) {
  extern  __shared__ int smem[]；
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  smem[threadIdx.y * blockDim.x + threadIdx.x] += a[idx * width + idy] * b[];
  smem[threadIdx.y * blockDim.x + threadIdx.x] += a[idx * width + idy] * b[];

}
int main(int argc, char* argv[]) {
  dim3 block(32, 16);
  int height = 1 << 11;
  int width = 1 << 11;
  int N = height * width;
  int width2 = 1 << 10;
  int n = width * width2;
  int n1 = height * width2;
  dim3 grid(width * width2 / block.x, height / block.y);
  int *a, *b, *c;
  a = (int*)malloc(sizeof(int)*N);
  hipMallocManaged((void**)&a, sizeof(int)*N);
  b = (int*)malloc(sizeof(int)*n);
  hipMallocManaged((void**)&a, sizeof(int)*n);
  hipMallocManaged((void**)&c, sizeof(int)*n1);
  matmul<<<grid, block, block.x * block.y * 2 * sizeof(int)>>>(a, b, target, height, width, width2, N);
  hipDeviceSynchronize();
  return 0;
}
