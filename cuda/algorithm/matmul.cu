
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__device__ int reduction(int a) {
  a += __shfl_xor(a, 16);
  a += __shfl_xor(a, 8);
  a += __shfl_xor(a, 4);
  a += __shfl_xor(a, 2);
  a += __shfl_xor(a, 1);
  return a;
}
void h_matmul(int *a, int* b,int* target, int height, int width, int width2, int n ) {
  for(int i = 0;i < height;++i) {
    for (int j = 0;j < width2;++j) {
      for (int k = 0;k < width;++k) {
        target[i * width2 + j] += a[i * height + k] * b[width2 * k + j];
      }
    }
  }
}
//thread should match target's size otherwith input's size 
//per = ceil(width / blockDim.x)
template<unsigned int per>
void __global__ matmul(int *a, int* b,int* target, int height, int width, int width2, int n) {
  //__shared__ int src[blockDim.y][blockDim.x + 1];
  __shared__ int src[32][32];
  __shared__ int tmp_target[32][32][per];
  int b_id = blockIdx.x % per;
  int idx = b_id * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idy < height && idx < width) {
    src[threadIdx.y][threadIdx.x] = a[idy * width + idx];
  }
  int pos = blockIdx.y * blockDim.x + threadIdx.x;
  idx = pos / blockDim.y;
  idy = pos % blockDim.y;
  //every thread deal blockDim.x datas
  int count = 0;
  for (int i = b_id;i < width; i+=blockDim.x) {
    tmp_target[blockIdx.y][threadIdx.x][count++] = src[idy][idx] * b[idx * width2 + i];
  }
  __syncthreads();
  int warp_id = pos % 32;
  int tmp = 0;
  for (int i = 0;i < count;++i) {
    tmp = tmp_target[warp_id][threadIdx.x][count];
    tmp = reduction(tmp);
    atomicAdd(target + idy * width2 + i*blockDim.x + b_id, tmp);
  }
}
int main(int argc, char* argv[]) {
  dim3 block(32, 32);
  int height = 1 << 10;
  int width = 1 << 10;
  int N = height * width;
  constexpr int width2 = 1 << 10;
  int n = width * width2;
  int n1 = height * width2;
  dim3 grid(width2 / block.x, height / block.y);
  int *a, *b, *c;
  a = (int*)malloc(sizeof(int)*N);
  hipMallocManaged((void**)&a, sizeof(int)*N);
  b = (int*)malloc(sizeof(int)*n);
  hipMallocManaged((void**)&b, sizeof(int)*n);
  c = (int*)malloc(sizeof(int)*n1);
  hipMallocManaged((void**)&c, sizeof(int)*n1);
  constexpr int per = (width2 + 32 - 1)/ 32;
  matmul<per><<<grid, block>>>(a, b, c, height, width, width2, N);
  hipDeviceSynchronize();
  return 0;
}
