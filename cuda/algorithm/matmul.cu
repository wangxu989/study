
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<ctime>
using namespace std;
__device__ int reduction(int a) {
  a += __shfl_xor(a, 16);
  a += __shfl_xor(a, 8);
  a += __shfl_xor(a, 4);
  a += __shfl_xor(a, 2);
  a += __shfl_xor(a, 1);
  return a;
}
void h_matmul(int *a, int* b,int* target, int height, int width, int width2, int n ) {
  for(int i = 0;i < height;++i) {
    for (int j = 0;j < width2;++j) {
      for (int k = 0;k < width;++k) {
        target[i * width2 + j] += a[i * height + k] * b[width2 * k + j];
      }
    }
  }
}
//thread should match target's size otherwith input's size 
//per = ceil(width / blockDim.x)
template<unsigned int step, unsigned per>
void __global__ matmul(int *a, int* b,int* target, int height, int width, int width2, int n) {
  //__shared__ int src[blockDim.y][blockDim.x + 1];
  __shared__ int src[32][32 + 1];
  __shared__ int tmp_target[32][32];
  int b_id = blockIdx.x % step;
  int idx = b_id * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idy < height && idx < width) {
    src[threadIdx.y][threadIdx.x] = a[idy * width + idx];
  }
  int pos = blockIdx.y * blockDim.x + threadIdx.x;
  int pos_x = pos / blockDim.y;
  int pos_y = pos % blockDim.y;
  //every thread deal width2/blockDim.x datas
  int count = 0;
  int tmp = 0;
  int warp_id = threadIdx.x%32;
  for (int i = b_id;i < width2; i += per) {
    tmp_target[pos_y][pos_x]  = src[pos_y][pos_x] * b[idx * width2 + i];
    tmp = tmp_target[threadIdx.y][threadIdx.x];
    __syncthreads();
    if(warp_id == 0) {
      atomicAdd(&target[idy * width2 + b_id], reduction(tmp));
    }
  }
}
int main(int argc, char* argv[]) {
  dim3 block(32, 32);
  int height = 1 << 10;
  int width = 1 << 10;
  int N = height * width;
  constexpr int width2 = 1 << 10;
  int n = width * width2;
  int n1 = height * width2;
  dim3 grid((width2 / block.x) * (width / block.x), height / block.y);
  int *a, *b, *c;
  a = (int*)malloc(sizeof(int)*N);
  hipMallocManaged((void**)&a, sizeof(int)*N);
  b = (int*)malloc(sizeof(int)*n);
  hipMallocManaged((void**)&b, sizeof(int)*n);
  c = (int*)malloc(sizeof(int)*n1);
  hipMallocManaged((void**)&c, sizeof(int)*n1);
  constexpr int per = 1024;
  constexpr int step = (width2 + per - 1)/ per;
  clock_t s, e;
  float time;
  hipEvent_t start,end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  s = clock();
  matmul<step, per><<<grid, block>>>(a, b, c, height, width, width2, N);
  hipEventRecord(end);
  hipEventSynchronize(end);
  e = clock();
  hipEventElapsedTime(&time, start ,end);
  cout << "gpu time is :" << time << "ms" << endl;
  cout << "gpu time is :" << e -s << "us" << endl;
  return 0;
}
