
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<ctime>
using namespace std;
__device__ int reduction(int a) {
  a += __shfl_xor(a, 16);
  a += __shfl_xor(a, 8);
  a += __shfl_xor(a, 4);
  a += __shfl_xor(a, 2);
  a += __shfl_xor(a, 1);
  return a;
}
void h_matmul(int *a, int* b,int* target, int height, int width, int width2, int n ) {
  for(int i = 0;i < height;++i) {
    for (int j = 0;j < width2;++j) {
      for (int k = 0;k < width;++k) {
        target[i * width2 + j] += a[i * height + k] * b[width2 * k + j];
      }
    }
  }
}
//thread should match target's size otherwith input's size 
//per = ceil(width / blockDim.x)
template<unsigned int step, unsigned int per>
void __global__ matmul(int *a, int* b,int* target, int height, int width, int width2, int n) {
  //__shared__ int src[blockDim.y][blockDim.x + 1];
  __shared__ int src[32][32];
  __shared__ int tmp[32][per];
  int b_id = blockIdx.x % step;
  int idx = b_id * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idy < height && idx < width) {
    src[threadIdx.y][threadIdx.x] = a[idy * width + idx];
    tmp[threadIdx.y][threadIdx.x] = b[idx * width2 + b_id + threadIdx.x * per];
  }
  //int pos = blockIdx.y * blockDim.x + threadIdx.x;
  //int pos_x = pos / blockDim.y;
  //int pos_y = pos % blockDim.y;
  //every thread deal width2/blockDim.x datas
  int c = 0;
  int warp_id = threadIdx.x%32;
  for (int i = 0;i < per;++i) {
    c  = src[threadIdx.y][threadIdx.x] * tmp[threadIdx.x][i];
    //tmp = tmp_target[threadIdx.y][threadIdx.x];
    //__syncthreads();
    if(warp_id == 0) {
      atomicAdd(&target[idy * width2 + b_id], reduction(c));
    }
  }
}
int main(int argc, char* argv[]) {
  dim3 block(32, 32);
  int height = 1 << 12;
  int width = 1 << 12;
  int N = height * width;
  constexpr int width2 = 1 << 12;
  int n = width * width2;
  int n1 = height * width2;
  constexpr int per = 32;
  constexpr int step = (width2 + per - 1)/ per;
  dim3 grid((width2 / per) * (width / block.x), height / block.y);
  int *a, *b, *c, *sum;
  a = (int*)malloc(sizeof(int)*N);
  hipMallocManaged((void**)&a, sizeof(int)*N);
  b = (int*)malloc(sizeof(int)*n);
  hipMallocManaged((void**)&b, sizeof(int)*n);
  c = (int*)malloc(sizeof(int)*n1);
  hipMallocManaged((void**)&c, sizeof(int)*n1);
  hipMallocManaged((void**)&sum, sizeof(int));
  clock_t s, e;
  float time;
  hipEvent_t start,end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  s = clock();
  matmul<step, per><<<grid, block>>>(a, b, c, height, width, width2, N);
  hipEventRecord(end);
  hipEventSynchronize(end);
  e = clock();
  hipEventElapsedTime(&time, start ,end);
  cout << "gpu time is :" << time << "ms" << endl;
  cout << "gpu time is :" << e -s << "us" << endl;
  int sum_elems = 0;
  hipMemcpy(&sum_elems, sum, sizeof(int), hipMemcpyDeviceToHost);
  cout << sum_elems << endl;
  return 0;
}
