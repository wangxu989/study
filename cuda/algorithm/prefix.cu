
#include <hip/hip_runtime.h>
#include<iostream>
#include<algorithm>
#include<ctime>
using namespace std;
template<unsigned int warp_size, unsigned int loop, typename T>
__global__ void prefix(T* begin, T* end, T* out, int n) {
  extern __shared__ T smem[];
  int idx = threadIdx.x;
  int laneid = idx % warp_size;
  int warp_idx = idx / warp_size;
  T acc_warp_sum = 0;
  for(int i = idx;i < n;i += blockDim.x) {
    T data = begin[i];
    T tmp = data;
    T t = 0;
#pragma unroll
    for(int j = 1;j < warp_size;j <<= 1) {
      t = __shfl_up(tmp, j);
      if(laneid >= j) {
        tmp += t;
      }
    }
    if(laneid == warp_size - 1) {
      smem[warp_idx] = tmp;
    }
    __syncthreads();
    T warp_tmp = 0;
    if(threadIdx.x < loop) {
      T warp_data = smem[threadIdx.x];
      warp_tmp = warp_data;
#pragma unroll
      for(int j = 1;j < warp_size;j <<= 1) {
        t = __shfl_up(warp_tmp, j);
        if(laneid >= j) {
          warp_tmp += t;
        }
      }
      smem[threadIdx.x] = warp_tmp;
    }
    __syncthreads();
    out[i] = (warp_idx > 0 ?smem[warp_idx - 1]:0) + tmp - data + acc_warp_sum;
    acc_warp_sum += smem[loop - 1];
  }
}
template<typename T>
void host_prefix(T*a ,int n) {
  int sum = 0;
  for(int i = 0;i < n;++i) {
    sum += a[i];
    a[i] = sum - a[i];
  }
}
typedef float dtype;
int main() {
  dtype *a;
  int n = 1<<20;
  n *= 100;
  dtype *b, *c;
  b = (dtype*)malloc(sizeof(dtype) * n);
  c = (dtype*)malloc(sizeof(dtype) * n);
  clock_t start,end;
  for(int i = 0;i < n;i++){b[i] = c[i] = random();}
  hipMalloc((void**)&a, sizeof(dtype) * n);
  hipMemcpy(a, b, sizeof(dtype)*n, hipMemcpyHostToDevice);
  start = clock();
  prefix<32, 32><<<1, 1024, 32*4>>>(a, a + n, a, n);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(b, a, sizeof(dtype)*n, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  //for_each(b, b + n, [](int a)->void{cout << a << " ";});
  cout << "\n";
  cout << "time:" << end - start << endl;
  start = clock();
  host_prefix(c, n);
  end = clock();
  cout << "time:" << end - start << endl;
  //for(int i = 0;i < n;++i) {
  //  if(b[i] != c[i]) {
  //    cout << "error: "<<b[i] << " not equal " <<c[i]<<"\n";
  //  }
  //}
  return 0;
}


