
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
typedef uint32_t RadixType;
__device__ unsigned int convert(float v) {
  // if v >= 0 v &= 2^31 else v &= 2^32 - 1,ensure anyif v >=0 or v < 0 by bitwise compare is vaild
  RadixType x = __float_as_int(v);
  RadixType mask = (x & 0x80000000) ? 0xffffffff : 0x80000000;
  return (v == v) ? (x ^ mask) : 0xffffffff;
}

__device__  float deconvert(RadixType v) {
  RadixType mask = (v & 0x80000000) ? 0x80000000 : 0xffffffff;
  return __int_as_float(v ^ mask);
}
__global__ void test() {
  float x = 1 << 20;
  printf("%f %u    %f",x , convert(x),deconvert(convert(x)));
}

int main() {
  dim3 block(1,1);
  dim3 grid(1,1);
  test<<<block,grid>>>();
  hipDeviceSynchronize();
  return 0;
}
