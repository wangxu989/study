#include "hip/hip_runtime.h"
#include<iostream>
#include<ctime>
#include<assert.h>
#include<algorithm>
#define EACH_THREAD 32
#define SIZE 512
using namespace std;
template<typename T>
__device__ unsigned int convert(T t) {
  assert(false);
}
template<>
__device__ __host__ unsigned int convert(float v) {
  // if v >= 0 v |= 2^31 else v = ~(v) - 1,ensure anyif v >=0 or v < 0 by bitwise compare is vaild
  unsigned int cmp = *reinterpret_cast<unsigned int*>(&v);
  unsigned int ret = (cmp & (1<<31)) ?  ~(cmp): (cmp | 0x80000000);
  return ret;
}

__device__ __host__ float deconvert(unsigned int v) {
  unsigned int tmp = (v & (1 << 31)) ?  (v ^ 0x80000000) : ~(v);
  return *reinterpret_cast<float*>(&tmp);
}

template<unsigned int each_thread>
void __global__ RadixSort(float* data, unsigned int* sort_tmp0,unsigned int* sort_tmp1, unsigned int n) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x * each_thread;
  unsigned int tid = threadIdx.x;
  if(idx >= n) {
    return ;
  }
  //bitwise push
  for (unsigned int bit = 0;bit < 32;bit++) {
      unsigned int mask = 1<<bit;
      unsigned int cnt0 = 0,cnt1 = 0;
      for(unsigned int i = tid;i < n;i += blockDim.x) {
        unsigned int elem = (bit == 0 ? convert(data[i]):sort_tmp0[i]);
        if((elem&mask) != 0) {
          sort_tmp1[cnt1 + tid] = elem;
          cnt1 += blockDim.x;
        }
        else {
          sort_tmp0[cnt0 + tid] = elem;
          cnt0 += blockDim.x;
        }
      }
      for (unsigned int i = 0;i < cnt1;i += blockDim.x) {
        sort_tmp0[cnt0 + i + tid] = sort_tmp1[i + tid];
      }
  }
  //merge
  __shared__ unsigned int min_value, min_tid;
  __shared__ unsigned int list_idx[512];
  unsigned int elem = 0xffffffff;
  list_idx[tid] = 0;
  __syncthreads();
  for(unsigned int i = 0;i < n;i++) {
    unsigned int x = (list_idx[tid] * blockDim.x) + tid;
    if(x < n) {
      elem = sort_tmp0[x];
    }
    else {
      elem = 0xffffffff;
    }
    if(tid == 0) {
      min_value = min_tid = 0xffffffff;
    }
    __syncthreads();
    atomicMin(&min_value,elem);
    __syncthreads();

    if(min_value == elem) {
      atomicMin(&min_tid, tid);
    }
    __syncthreads();
    if(min_tid == tid) {
      list_idx[tid]++;
      data[i] = deconvert(min_value);
    }
    __syncthreads();
  }
}

void RadixSortHost(float *v,unsigned int size) {
  unsigned int sort_tmp0[size],sort_tmp1[size];
  for(int bit = 0;bit < 32;bit++) {
    unsigned int mask = 1 << bit;
    unsigned int cnt0 = 0,cnt1 = 0;
    for (int i = 0;i < size;i++) {
      unsigned int elem = ((bit == 0) ?convert(v[i]):sort_tmp0[i]);
      if((elem & mask) != 0) {
        sort_tmp1[cnt1++] = elem;
      }
      else {
        sort_tmp0[cnt0++] = elem;
      }
    }
    for(int i = 0;i < cnt1;i++) {
      sort_tmp0[cnt0 + i] = sort_tmp1[i];
    }
  }
  for(int i = 0;i < size;i++) {
    v[i] = deconvert(sort_tmp0[i]);
  }
}
__shared__ unsigned int sort_tmp0[SIZE * EACH_THREAD];
__shared__ unsigned int sort_tmp1[SIZE * EACH_THREAD];

int main() {
  int N = SIZE * EACH_THREAD;
  float a[N],b[N];
  auto init = [](auto*a ,unsigned int size)->void {
    for(int i = 0;i < size;i++) {
      a[i] = pow(-1,i) * (random()%1000);
    }
  };
  init(a, N);
  float *a_dev;
  clock_t start ,end;
  unsigned int *sort_tmp0, *sort_tmp1;
  dim3 block(512,1);
  dim3 grid( (N + block.x -1) / block.x /EACH_THREAD , 1);
  hipMalloc((void**)&a_dev, sizeof(float)*N);
  hipMalloc((void**)&sort_tmp0, sizeof(unsigned int)*N);
  hipMalloc((void**)&sort_tmp1, sizeof(unsigned int)*N);
  hipMemcpy(a_dev, a, sizeof(float)*N,hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  start = clock();
  RadixSort<EACH_THREAD><<<grid,block>>>(a_dev, sort_tmp0, sort_tmp1, N);
  hipDeviceSynchronize();
  end = clock();
  cout << "gpu time:" << end - start << endl;
  hipMemcpy(b, a_dev, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0;i < N;i++) {
   // cout << b[i] <<" ";
  }
  cout << endl;
  RadixSortHost(a,N);
  for (int i = 0;i < N;i++) {
    if(a[i] != b[i]) {
      //printf("error: index%u gpu:%f cpu:%f\n",i,b[i], a[i]);
    }
  }

  return 0;
}
