#include "hip/hip_runtime.h"
#include<iostream>
#include<ctime>
#include<assert.h>
#include<algorithm>
#define EACH_THREAD 256
#define SIZE 1024
using namespace std;
template<typename T>
__device__ unsigned int convert(T t) {
  assert(false);
}
template<>
__device__ __host__ unsigned int convert(float v) {
  // if v >= 0 v |= 2^31 else v = ~(v) - 1,ensure anyif v >=0 or v < 0 by bitwise compare is vaild
  unsigned int cmp = *reinterpret_cast<unsigned int*>(&v);
  unsigned int ret = (cmp & (1<<31)) ?  ~(cmp): (cmp | 0x80000000);
  return ret;
}

__device__ __host__ float deconvert(unsigned int v) {
  unsigned int tmp = (v & (1 << 31)) ?  (v ^ 0x80000000) : ~(v);
  return *reinterpret_cast<float*>(&tmp);
}

template<unsigned int each,unsigned int size>
void __global__ RadixSort(float* data, unsigned int n) {
  //unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x * each_thread;
  unsigned int tid = threadIdx.x;
  //bitwise push
  unsigned int local[each*2];
  for(unsigned int i = 0;i < each;++i) {
    //if(tid + i * blockDim.x < n) {
      local[i] = convert(data[tid + i * blockDim.x]);
    //}
  }
  __syncthreads();
  for (unsigned int bit = 0;bit < 32;bit++) {
      unsigned int mask = 1<<bit;
      unsigned int cnt[2] = {0,0};
      for(unsigned int i = 0;i < each;++i) {
        unsigned int elem = local[i];
        unsigned int index_type = (mask&elem)>>bit;
        local[cnt[index_type] + index_type * each] = elem;
        cnt[index_type]++;
      }
      for (unsigned int i = 0;i < cnt[1];++i) {
        local[cnt[0] + i] = local[i + each];
      }
  }
  //merge
 __shared__ unsigned int min_value, min_tid;
 __shared__ unsigned int list_idx[size];
 unsigned int elem = 0xffffffff;
 list_idx[tid] = 0;
 __syncthreads();
 for(unsigned int i = 0;i < n;i++) {
   elem = 0xffffffff;
   if(list_idx[tid] < each) {
     elem = local[list_idx[tid]];
   }
   __syncthreads();
   min_value = min_tid = 0xffffffff;
   atomicMin(&min_value,elem);
   __syncthreads();
   if(min_value == elem) {
     atomicMin(&min_tid, tid);
   }
   __syncthreads();
   if(min_tid == tid) {
     list_idx[tid]++;
     data[i] = deconvert(min_value);
   }
 }
}

void RadixSortHost(float *v,unsigned int size) {
  unsigned int sort_tmp0[size],sort_tmp1[size];
  for(int bit = 0;bit < 32;bit++) {
    unsigned int mask = 1 << bit;
    unsigned int cnt0 = 0,cnt1 = 0;
    for (int i = 0;i < size;i++) {
      unsigned int elem = ((bit == 0) ?convert(v[i]):sort_tmp0[i]);
      if((elem & mask) != 0) {
        sort_tmp1[cnt1++] = elem;
      }
      else {
        sort_tmp0[cnt0++] = elem;
      }
    }
    for(int i = 0;i < cnt1;i++) {
      sort_tmp0[cnt0 + i] = sort_tmp1[i];
    }
  }
  for(int i = 0;i < size;i++) {
    v[i] = deconvert(sort_tmp0[i]);
  }
}

int main() {
  int N = SIZE * EACH_THREAD;
  float a[N],b[N];
  auto init = [](auto*a ,unsigned int size)->void {
    for(int i = 0;i < size;i++) {
      a[i] = pow(-1,i) * (random()%1000);
    }
  };
  init(a, N);
  float *a_dev;
  clock_t start ,end;
  dim3 block(SIZE,1);
  dim3 grid(1, 1);
  hipMalloc((void**)&a_dev, sizeof(float)*N);
  hipMemcpy(a_dev, a, sizeof(float)*N,hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  start = clock();
  RadixSort<EACH_THREAD,SIZE><<<grid,block>>>(a_dev, N);
  hipDeviceSynchronize();
  end = clock();
  cout << "gpu time:" << end - start << endl;
  hipMemcpy(b, a_dev, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0;i < N;i++) {
   // cout << b[i] <<" ";
  }
  cout << endl;
  start = clock();
  RadixSortHost(a,N);
  end = clock();
  cout << "cpu time:" << end - start << endl;
  for (int i = 0;i < N;i++) {
    if(a[i] != b[i]) {
      printf("error: index%u gpu:%f cpu:%f\n",i,b[i], a[i]);
    }
  }

  return 0;
}
