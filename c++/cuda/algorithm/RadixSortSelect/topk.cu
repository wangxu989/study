
#include <hip/hip_runtime.h>
#include<iostream>
#include<assert.h>
#include<algorithm>
#define N 10000
#define K 30
#define RadixBit 2
#define RadixSize 4//2 ^ RadixBit
#define RadixMask 3
using namespace std;

__device__ int get_Laneid() {
  int laneId;
  asm("mov.s32 %0, %%laneid;" : "=r"(laneId) );
  return laneId;
}

__device__ unsigned int getBitwise(unsigned int val,unsigned int pos,unsigned int bits) {
  unsigned int m = (1 << bits) - 1u;
  return (val >> pos) & m; 
}
__device__ unsigned int setBitwise(unsigned int val, unsigned int insertval,unsigned int pos, unsigned int bits) {
  unsigned int m = 1 << bits - 1u;
  insertval <<= pos;
  m <<= pos;
  return (val & ~m) | insertval;
}
__device__ void countingMaskedval(int *count,
    unsigned int *data, 
    unsigned int desired, 
    unsigned int mask,
    int pos,
    unsigned int *smem, 
    unsigned int n) {
  for(int i = 0;i < RadixSize;i++) {
    count[i] = 0;
  }
  //initializer
  if(threadIdx.x < RadixSize) {
    smem[threadIdx.x] = 0;
  }
  __syncthreads();

  for(unsigned int i = threadIdx.x;i < n;i += blockDim.x) {
    unsigned int valbit = getBitwise(data[i], pos, RadixBit);
    bool hasval = ((data[i] & mask) == desired);
    for(int j = 0;j < RadixSize;j++) {
      bool vote = hasval && (valbit == j);
      count[j] += __popc(__ballot_sync(0xffffffff,vote));
    }

  }
  if(get_Laneid() == 0) {
    for(int i = 0;i < RadixSize;i++) {
      atomicAdd(smem + i, count[i]);
    }
  }
  __syncthreads();
  for(int i = 0;i < RadixSize;i++) {
    count[i] = smem[i];
  }
  __syncthreads();

}
unsigned int __device__ findPattern(unsigned int *data,unsigned int *smem, unsigned int desired, unsigned int mask, unsigned int n) {
  if (threadIdx.x < 2) {
    smem[threadIdx.x] = 0;
  }
  __syncthreads();
  for(int i = threadIdx.x;i < n;i += blockDim.x) {
    if((data[i] & mask) == desired) {
      smem[0] = 1;
      smem[1] = data[i];
    }
  }
  __syncthreads();
  unsigned int found = smem[0];
  unsigned int val = smem[1];
  __syncthreads();
  if(found == 1) {
    //one thread find the unique data
    //and every return this value
    //printf("%u ",val);
    return val;
  }
  assert(false);
  //do not find the data
  printf("%u ",val);
  return 0;
}
void __device__ RadixSelect(unsigned int *data,unsigned int n,unsigned int *smem, int k, unsigned int *topk) {
  //every thread has mask,desired,count to deal N/blockDim.x datas
  int count[RadixSize];
  unsigned int desired = 0;
  unsigned int mask = 0;
  int ktofind = k;
  unsigned int ret;
  for(int pos = sizeof(unsigned int)*8 - RadixBit;pos >=0;pos -= RadixBit) {
    countingMaskedval(count, data, desired, mask, pos, smem, n);
    auto find_unique = [&](int i, int counts) {
      if(counts == 1 && ktofind == 1) {
        desired = setBitwise(desired, i, pos, RadixBit);
        mask = setBitwise(mask, RadixMask, pos, RadixBit);
        //in every thread's head
        //now we know somedata & mask = desired is unique,and we will find this data;
        *topk = findPattern(data, smem, desired, mask, n);
        return true;
      }
      return false;
    };
    auto find_non_unique = [&](int i, int counts) {
      if(counts >= ktofind) {
        desired = setBitwise(desired, i, pos, RadixBit);
        mask = setBitwise(mask, RadixMask, pos, RadixBit);
        //continue find and the topk is in which & mask = desired
        return true;
      }
      //continue find 
      ktofind -= counts;
      return false;
    };
    for(int i = RadixSize - 1;i >= 0;i--) {
      int c = count[i];
      if(find_unique(i, c)) {
        return;
      }
      if(find_non_unique(i, c)) {
        //continue
        break;
      }
    }

  }
  //the topk has some same data,we return the same data is ok
  *topk = desired;
}

void __global__ findtopK(unsigned int* data, unsigned int n, unsigned int *topk) {
  __shared__ unsigned int smem[64];
  RadixSelect(data, n, smem, blockIdx.x + 1, topk + blockIdx.x);
}
int main() {
  unsigned int data[N],*data_dev;
  unsigned int topk[K],*topk_dev;
  for(int i = 0;i < N;i++) {
    data[i] = random()%1000;
  }
  hipMalloc((void**)&data_dev, sizeof(data));
  hipMalloc((void**)&topk_dev, sizeof(topk));
  hipMemcpy(data_dev, data ,sizeof(data), hipMemcpyHostToDevice);
  findtopK<<<K,1024>>>(data_dev, N, topk_dev);
  hipMemcpy(topk, topk_dev ,sizeof(topk), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  sort(data, data + N, [](unsigned int a,unsigned int b) {
        return a > b;
      });
  for(int i = 0;i < K;i++) {
    if(data[i] != topk[i]) {
      cout << "faild !!!" << "index: " << i << "cpu:" << data[i] << " gpu: " << topk[i] <<endl; 
    }
  }
  return 0;
}
