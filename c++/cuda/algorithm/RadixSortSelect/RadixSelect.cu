#include "hip/hip_runtime.h"
#include<stdio.h>
#include<algorithm>
#include<iostream>
#include"./findPattern.h"
#include<ctime>
#define RADIX_SIZE 4
#define RADIX_BITS 2
#define RADIX_MASK 3
#define K 1
#define order true

 // bool order = true ;max topk else min topk
using namespace std;

static inline __device__ void gpuAtomicAdd(int32_t *address, int32_t val) {
  atomicAdd(address, val);
}
__device__ __forceinline__ int getLaneId() {
#if defined(__HIP_PLATFORM_HCC__)
  return __lane_id();
#else
  int laneId;
  asm("mov.s32 %0, %%laneid;" : "=r"(laneId) );
  return laneId;
#endif
}

__device__ __forceinline__ unsigned int ACTIVE_MASK()
{
#ifndef __HIP_PLATFORM_HCC__
  return __activemask();
#else
  // will be ignored anyway
  return 0xffffffff;
#endif
}


#if defined(__HIP_PLATFORM_HCC__)
__device__ __forceinline__ unsigned long long int WARP_BALLOT(int predicate)
{
  return __ballot(predicate);
}
#else
__device__ __forceinline__ unsigned int WARP_BALLOT(int predicate, unsigned int mask = 0xffffffff)
{
#ifndef __HIP_PLATFORM_HCC__
  return __ballot_sync(mask, predicate);
#else
  return __ballot(predicate);
#endif
}
#endif


template<typename T>
struct Bitfield{};


template<>
struct Bitfield<unsigned int> {
  static __device__ __forceinline__
    unsigned int getBitfield(unsigned int val, int pos, int len) {
#if defined(__HIP_PLATFORM_HCC__)
      pos &= 0xff;
      len &= 0xff;

      unsigned int m = (1u << len) - 1u;
      return (val >> pos) & m;
#else
      unsigned int ret;
      asm("bfe.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(val), "r"(pos), "r"(len));
      return ret;
#endif
    }

  static __device__ __forceinline__
    unsigned int setBitfield(unsigned int val, unsigned int toInsert, int pos, int len) {
#if defined(__HIP_PLATFORM_HCC__)
      pos &= 0xff;
      len &= 0xff;

      unsigned int m = (1u << len) - 1u;
      toInsert &= m;
      toInsert <<= pos;
      m <<= pos;

      return (val & ~m) | toInsert;
#else
      unsigned int ret;
      asm("bfi.b32 %0, %1, %2, %3, %4;" :
          "=r"(ret) : "r"(toInsert), "r"(val), "r"(pos), "r"(len));
      return ret;
#endif
    }
};

template <typename scalar_t,
         typename bitwise_t,
         typename index_t,
         typename CountType,
         int RadixSize,
         int RadixBits>
         __device__ void countRadixUsingMask(
             CountType counts[RadixSize],
             CountType* smem,
             bitwise_t desired,
             bitwise_t desiredMask,
             int radixDigitPos,
             index_t sliceSize,
             index_t withinSliceStride,
             scalar_t* data) {
           // Clear out per-thread counts from a previous round
#pragma unroll
           for (int i = 0; i < RadixSize; ++i) {
             counts[i] = 0;
           }

           if (threadIdx.x < RadixSize) {
             smem[threadIdx.x] = 0;
           }
           __syncthreads();

           // Scan over all the data. Upon a read, the warp will accumulate
           // counts per each digit in the radix using warp voting.
           for (index_t i = threadIdx.x; i < sliceSize; i += blockDim.x) {
             bitwise_t val =
               TopKTypeConfig<scalar_t>::convert(doLdg(&data[i * withinSliceStride]));

             bool hasVal = ((val & desiredMask) == desired);
             bitwise_t digitInRadix =
               Bitfield<bitwise_t>::getBitfield(val, radixDigitPos, RadixBits);

#pragma unroll
             for (uint32_t j = 0; j < RadixSize; ++j) {
               bool vote = hasVal && (digitInRadix == j);
#if defined(__HIP_PLATFORM_HCC__)
               counts[j] += __popcll(WARP_BALLOT(vote));
#else
               counts[j] += __popc(WARP_BALLOT(vote, ACTIVE_MASK()));
#endif
             }
           }

           // Now, for each warp, sum values
           //first thread of warp
           if (getLaneId() == 0) {
#pragma unroll
             for (uint32_t i = 0; i < RadixSize; ++i) {
               gpuAtomicAdd(&smem[i], counts[i]);
             }
           }

           __syncthreads();

           // For each thread, read in the total counts
#pragma unroll
           for (uint32_t i = 0; i < RadixSize; ++i) {
             counts[i] = smem[i];
             //printf("%u  ",counts[i]);
           }

           __syncthreads();
         }

template <typename scalar_t, typename bitwise_t, typename index_t, bool Order>
__device__ void radixSelect(
    scalar_t* data,
    index_t k,
    index_t sliceSize,
    index_t withinSliceStride,
    int* smem,
    scalar_t* topK) {
  // Per-thread buckets into which we accumulate digit counts in our
  // radix
  int counts[RADIX_SIZE];

  // We only consider elements x such that (x & desiredMask) == desired
  // Initially, we consider all elements of the array, so the above
  // statement is true regardless of input.
  bitwise_t desired = 0;
  bitwise_t desiredMask = 0;

  // We are looking for the top kToFind-th element when iterating over
  // digits; this count gets reduced by elimination when counting
  // successive digits
  int kToFind = k;

  // We start at the most signific,ant digit in our radix, scanning
  // through to the least significant digit
#pragma unroll
  for (int digitPos = sizeof(scalar_t) * 8 - RADIX_BITS; digitPos >= 0;
       digitPos -= RADIX_BITS) {
    // Count radix distribution for the current position and reduce
    // across all threads
    countRadixUsingMask<
        scalar_t,
        bitwise_t,
        index_t,
        int,
        RADIX_SIZE,
        RADIX_BITS>(
        counts,
        smem,
        desired,
        desiredMask,
        digitPos,
        sliceSize,
        withinSliceStride,
        data);

    auto found_unique = [&](int i, int count) -> bool {
      /* All threads have the same value in counts here, so all */
      /* threads will return from the function. */
      if (count == 1 && kToFind == 1) {
        /* There is a unique answer. */
        desired =
            Bitfield<bitwise_t>::setBitfield(desired, i, digitPos, RADIX_BITS);
        desiredMask = Bitfield<bitwise_t>::setBitfield(
            desiredMask, RADIX_MASK, digitPos, RADIX_BITS);

        /* The answer is now the unique element v such that: */
        /* (v & desiredMask) == desired */
        /* However, we do not yet know what the actual element is. We */
        /* need to perform a search through the data to find the */
        /* element that matches this pattern. */
        *topK = findPattern<scalar_t, bitwise_t, index_t>(
            (scalar_t*)smem,
            data,
            sliceSize,
            withinSliceStride,
            desired,
            desiredMask);
        return true;
      }
      return false;
    };
    auto found_non_unique = [&](int i, int count) -> bool {
      if (count >= kToFind) {
        desired =
            Bitfield<bitwise_t>::setBitfield(desired, i, digitPos, RADIX_BITS);
        desiredMask = Bitfield<bitwise_t>::setBitfield(
            desiredMask, RADIX_MASK, digitPos, RADIX_BITS);

        /* The top-Kth element v must now be one such that: */
        /* (v & desiredMask == desired) */
        /* but we haven't narrowed it down; we must check the next */
        /* least-significant digit */
        return true;
      }
      kToFind -= count;
      return false; // continue the loop
    };

    // All threads participate in the comparisons below to know the
    // final result
    if (Order) {
      // Process in descending order
#pragma unroll
      for (int i = RADIX_SIZE - 1; i >= 0; --i) {
        int count = counts[i];
        //
        if (found_unique(i, count)) {
          return;
        }
        if (found_non_unique(i, count)) {
          break;
        }
      }
    } else {
      // Process in ascending order
#pragma unroll
      for (int i = 0; i < RADIX_SIZE; ++i) {
        int count = counts[i];
        if (found_unique(i, count)) {
          return;
        }
        if (found_non_unique(i, count)) {
          break;
        }
      }
    }
  } // end digitPos for

  // There is no unique result, but there is a non-unique result
  // matching `desired` exactly
  *topK = TopKTypeConfig<scalar_t>::deconvert(desired);
}

__global__ void __test(float *data,unsigned int sliceSize, float *topk) {
    unsigned int withinSliceStride = 1;
    __shared__  int smem[64];
    radixSelect<float, unsigned int, unsigned int,order>(data, blockIdx.x + 1, sliceSize, withinSliceStride, smem, topk + blockIdx.x);
}
//__shared__ int smem[2];
int main() {
  int N = 1024;
  float *data_dev,data[N];
  float topk[K],*topk_dev;
  for(int i = 0;i < N;i++) {
    data[i] = rand()%1000;
  }
  data[N - 1] = 10000;
  hipMalloc((void**)&topk_dev, sizeof(float)*K);
  hipMalloc((void**)&data_dev, sizeof(float)*N);
  hipMemcpy(data_dev,data,sizeof(data), hipMemcpyHostToDevice);
  clock_t start ,end;
  start = clock();
  __test<<<K,1024>>>(data_dev, N, topk_dev);
  hipDeviceSynchronize();
  end = clock();
  cout <<"time gpu:" << end - start << "us"<<endl;
  hipMemcpy(topk,topk_dev,sizeof(float)*K, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  start = clock();
  sort(data,data + N, [](float a, float b) {
        return a > b;
      });
  end = clock();
  cout <<"time cpu:" << end - start << "us"<<endl;
  hipMemcpy(&topk,topk_dev,sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0;i < K;i++) {
    if (data[i] != topk[i]) {
        cout <<"index:" << i <<"    "<< "failed !!!" << "cpu:" << data[i] << "gpu:" << topk[i] << endl;
    }
  }
  cout <<data[0] <<endl;
  return 0;
}
