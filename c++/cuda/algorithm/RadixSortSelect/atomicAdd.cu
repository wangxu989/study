
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void add(uint32_t *ret){
        uint32_t idx = threadIdx.x;
        __shared__ uint32_t sum;
        if(idx == 0) {
                sum = 0;
        }
        __syncthreads();
        atomicAdd(&sum,idx);
        __syncthreads();
        if(idx == 0) {
                *ret = sum;

        }
}
int main() {
        dim3 block(64,1);
        dim3 grid(1,1);
        uint32_t *ret;
        hipMalloc((void**)&ret, sizeof(uint32_t));
        add<<<grid, block>>>(ret);
        hipDeviceSynchronize();
        uint32_t ans = 0;
        hipMemcpy(&ans, ret, sizeof(uint32_t) ,hipMemcpyDeviceToHost);
        cout <<"gpu ans:" << ans << endl;
        cout <<"cpu ans:" << (0 + 63)*64/2 << endl;
}

