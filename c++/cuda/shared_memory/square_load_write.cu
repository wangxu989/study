
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
#define R 32
#define C 32
//#define BY_R

__global__ void by_row(int* data) {
  __shared__ int cache[R][C];
  unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
  cache[threadIdx.y][threadIdx.x] = idx;
  __syncthreads();
  data[idx] = cache[threadIdx.y][threadIdx.x];
}
__global__ void by_column(int* data) {
  __shared__ int cache[R][C];
  unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
  cache[threadIdx.x][threadIdx.y] = idx;
  __syncthreads();
  data[idx] = cache[threadIdx.x][threadIdx.y];
}

int main() {
  clock_t start, end;
  int *a_dev;
  dim3 block(R,C);
  hipMalloc((void**)&a_dev, sizeof(R*C));
  start = clock();
#ifdef BY_R
  by_row<<<1,block>>>(a_dev);
  cout << "gpu by_row    ";
#else
  by_column<<<1,block>>>(a_dev);
  cout << "gpu by_column    ";
#endif
  hipDeviceSynchronize();
  end = clock();
  cout << end - start << "us" << endl;
  return 0;
}
