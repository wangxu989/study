
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
#define Bits 20
#define BlockDimx 512
__device__ float out;
template<unsigned int  BlockSize, typename T>
__global__ void add(T *in_data, unsigned int N) {
  __shared__ T smem[BlockSize];
  T tmp_val = T(0);
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x*4;
  if (idx +  blockDim.x * 3 < N) {
    T a1 = in_data[idx];
    T a2 = in_data[idx + blockDim.x];
    T a3 = in_data[idx + blockDim.x * 2];
    T a4 = in_data[idx + blockDim.x * 3];
    tmp_val = a1 + a2 + a3 + a4;
  }
  smem[threadIdx.x] = tmp_val;
  __syncthreads();

  if(BlockSize >= 1024 && threadIdx.x < 512) {
    smem[threadIdx.x] += smem[threadIdx.x + 512];
  }
  __syncthreads();


  if(BlockSize >= 512 && threadIdx.x < 256) {
    smem[threadIdx.x] += smem[threadIdx.x + 256];
  }
  __syncthreads();
  if(BlockSize >= 256 && threadIdx.x < 128) {
    smem[threadIdx.x] += smem[threadIdx.x + 128];
  }
  __syncthreads();
  if(BlockSize >= 128 && threadIdx.x < 64) {
    smem[threadIdx.x] += smem[threadIdx.x + 64];
  }
  __syncthreads();
  if(threadIdx.x < 32) {
    volatile T* vmem = smem;
    vmem[threadIdx.x] += vmem[threadIdx.x + 32];
    vmem[threadIdx.x] += vmem[threadIdx.x + 16];
    vmem[threadIdx.x] += vmem[threadIdx.x + 8];
    vmem[threadIdx.x] += vmem[threadIdx.x + 4];
    vmem[threadIdx.x] += vmem[threadIdx.x + 2];
    vmem[threadIdx.x] += vmem[threadIdx.x + 1];
  }
  if(threadIdx.x == 0) {
    atomicAdd(&out, smem[0]);   
  }
}

int main(int argc ,char *argv[]) {
  unsigned int N = 1 << Bits;
  dim3 block(BlockDimx,1);
  dim3 grid((N + BlockDimx -1) / BlockDimx / 4, 1);
  float in[N], *in_dev, t = 0;
  clock_t start ,end;
  auto init = [](auto *in ,unsigned int size)->void {
    for(int i = 0;i < size;++i) {
      in[i] = 1;
    }
  };
  init(in, N);
  hipMalloc((void**)&in_dev, sizeof(in));
  hipMemcpy(in_dev, in, sizeof(in), hipMemcpyHostToDevice);
  start = clock();
  add<BlockDimx><<<grid, block>>>(in_dev, N);
  hipDeviceSynchronize();
  end = clock();
  float ans;
  hipMemcpyFromSymbol(&ans, HIP_SYMBOL(out), sizeof(float));
  hipDeviceSynchronize();
  printf("%f\n", ans);
  printf("%d\n", block.x * grid.x * 4);
  cout <<"gpu time: "<<end -start<<endl;
  return 0;
}
