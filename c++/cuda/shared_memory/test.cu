
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void test(float *data) {
  unsigned int tid = threadIdx.x;
  if(tid < 32) {
    volatile float *in = data;
    in[tid] += in[tid + 32];
    in[tid] += in[tid + 16];
    in[tid] += in[tid + 8];
    in[tid] += in[tid + 4];
    in[tid] += in[tid + 2];
    in[tid] += in[tid + 1];
  }
}
int main() {
  float in[64];
  for(int i = 0;i < 64;++i) {
    in[i] = 1;
  }
  float *in_dev;
  hipMalloc((void**)&in_dev, sizeof(in));
  hipMemcpy(in_dev, in , sizeof(in), hipMemcpyHostToDevice);
  dim3 block(32,1);
  dim3 grid(1,1);
  test<<<grid, block>>>(in_dev);
  hipMemcpy(in, in_dev , sizeof(in), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int i = 0;i < 64;++i) {
    cout << in[i] <<"  ";
  }
  return 0;
}
