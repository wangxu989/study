
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int main() {
  hipSharedMemConfig config;
  hipDeviceGetSharedMemConfig(&config);
  switch(config) {
    case hipSharedMemBankSizeDefault:
      cout << "default SharedConfig" << endl;
      break;
    case hipSharedMemBankSizeFourByte:
      cout << "fourbyte SharedConfig" << endl;
      break;
    case hipSharedMemBankSizeEightByte:
      cout << "eightbyte SharedConfig" << endl;
      break;
  }
  return 0;
}
