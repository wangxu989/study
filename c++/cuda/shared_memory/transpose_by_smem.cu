
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
template<unsigned int DIMX, unsigned int DIMY, typename T>
__global__ void transpose(T *in_data, T *out_data, unsigned int nx, unsigned int ny) {
  //padding = 2
  __shared__ T tile[DIMY][DIMX*2 + 2];
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x * 2;
  unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if(idx + blockDim.x < nx && idy < ny) {
    tile[threadIdx.y][threadIdx.x] = in_data[idy * nx + idx];
    tile[threadIdx.y][threadIdx.x + blockDim.x] = in_data[idy * nx + idx + blockDim.x];
    __syncthreads();
    unsigned int posB = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int column = posB / blockDim.y;
    unsigned int row = posB % blockDim.y;
    idx = column + blockDim.x * blockIdx.x * 2;
    idy = row + blockDim.y * blockIdx.y;
    out_data[idx * ny + idy] = tile[row][column];
    out_data[(idx + blockDim.x) * ny + idy] = tile[row][column + blockDim.x];
  }
}
template<typename T>
void transposeHost(T *in, T* out, unsigned int nx, unsigned int ny) {
  for(int i = 0;i < nx;++i) {
    for(int j = 0;j < ny;++j) {
      out[i * ny + j] = in[j * nx + i];
    }
  }
}
int main(int argc, char *argv[]) {
  unsigned int nx = 1 << 9;
  unsigned int ny = 1 << 9;
  constexpr unsigned int blockx = 32;
  constexpr unsigned int blocky = 32;
  clock_t start, end;
  int in[nx * ny], out[nx * ny], *in_dev, *out_dev;
  auto init = [](auto*in ,unsigned int size)->void {
    for(int i = 0;i < size;++i) {
      in[i] = random()%1000;
    }
  };
  init(in, nx * ny);
  hipMalloc((void**)&in_dev, sizeof(in));
  hipMalloc((void**)&out_dev, sizeof(in));
  hipMemcpy(in_dev, in ,sizeof(in), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  transposeHost(in, out, nx, ny);
  dim3 block(blockx, blocky);
  dim3 grid((nx + blockx - 1) / blockx / 2, (ny + blocky - 1) / blocky);
  start = clock();
  transpose<blockx, blocky><<<grid, block>>>(in_dev, out_dev, nx,ny);
  hipDeviceSynchronize();
  end = clock();
  cout <<" gpu time: " << end - start<<endl;
  hipMemcpy(in, out_dev,sizeof(in), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(in_dev);
  hipFree(out_dev);
  int n = 0;
  for (int i = 0;i < nx * ny;++i) {
    if(out[i] != in[i]) {
      n++;
    }
  }
  cout << n << endl;
  return 0;
}
