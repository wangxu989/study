
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
//test file product  on GPU
template<unsigned int blocksize, typename T>
__global__ void prob(T* a, T* b,unsigned int n) {
  unsigned int tid = threadIdx.x;
  unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x*8;
  T *data = a + blockDim.x*blockIdx.x*8;
  //增加存储效率
  if(idx  + blockDim.x*7 < n) {
    a[idx] *= a[idx + blockDim.x];
    a[idx] *= a[idx + blockDim.x*2];
    a[idx] *= a[idx + blockDim.x*3];
    a[idx] *= a[idx + blockDim.x*4];
    a[idx] *= a[idx + blockDim.x*5];
    a[idx] *= a[idx + blockDim.x*6];
    a[idx] *= a[idx + blockDim.x*7];
  }
  __syncthreads();
  //规约
  if(blocksize >= 1024 && tid < 512) {
    data[tid] *= data[tid + 512];
  }
  __syncthreads();
  if(blocksize >= 512 && tid < 256) {
    data[tid] *= data[tid + 256];
  }
  __syncthreads();
  if(blocksize >= 256 && tid < 128) {
    data[tid] *= data[tid + 128];
  }
  __syncthreads();
  if(blocksize >= 128 && tid < 64) {
    data[tid] *= data[tid + 64];
  }
  __syncthreads();

  if(tid < 32) {
    volatile T *vmem = data;
    vmem[tid] *= vmem[tid + 32];
    vmem[tid] *= vmem[tid + 16];
    vmem[tid] *= vmem[tid + 8];
    vmem[tid] *= vmem[tid + 4];
    vmem[tid] *= vmem[tid + 2];
    vmem[tid] *= vmem[tid + 1];
  }
  if(tid == 0) {b[blockIdx.x] = data[0];}
}
int main() {
  unsigned int N = 1<<20;
  int SIZE = 512;
  dim3 block(SIZE,1);
  dim3 grid((block.x + N - 1)/block.x,1);
  float a[N],b[grid.x];
  for (int i = 0;i < N;i++) {
    a[i] = 1;
  }
  float *a_dev, *b_dev;
  hipMalloc((float**)&a_dev, sizeof(float)*N);
  hipMalloc((float**)&b_dev, sizeof(float)*grid.x);
  hipMemcpy(a_dev, a, sizeof(float)*N, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  switch(SIZE) {
    case 512:
    prob<512><<<grid.x/8,block>>>(a_dev, b_dev, N);
    break;
  }
  hipMemcpy(b, b_dev, sizeof(float)*grid.x, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  float ans = 1.0f;
  for(int i = 0;i < grid.x;i++) {
    ans += b[i];
  }
  cout<<ans<<endl;
  return 0;
}
