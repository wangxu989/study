
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
#define O3
template<unsigned int BlockSize>
__global__ void add(int*a,int*b,unsigned int n) {
  unsigned int tid = threadIdx.x;
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;
  int *idata = a + blockIdx.x * blockDim.x*8;
  if(idx + 7*blockDim.x < n) {
    a[idx] += a[idx + blockDim.x];
    a[idx] += a[idx + 2*blockDim.x];
    a[idx] += a[idx + 3*blockDim.x];
    a[idx] += a[idx + 4*blockDim.x];
    a[idx] += a[idx + 5*blockDim.x];
    a[idx] += a[idx + 6*blockDim.x];
    a[idx] += a[idx + 7*blockDim.x];
  }
  //if(idx  + blockDim.x < n) a[idx] += a[idx + blockDim.x];
  __syncthreads();
  if(BlockSize >= 1024 && tid < 512) {
    idata[tid] += idata[tid + 512];
  }
  __syncthreads();
  if(BlockSize >= 512 && tid < 256) {
    idata[tid] += idata[tid + 256];
  }
  __syncthreads();
  if(BlockSize >= 256 && tid < 128) {
    idata[tid] += idata[tid + 128];
  }
  __syncthreads();
  if(BlockSize >= 128 && tid < 64) {
    idata[tid] += idata[tid + 64];
  }
  __syncthreads();
  //unfold warp
  if(tid < 32) {
    volatile int *vmem = idata;
    vmem[tid] += vmem[tid + 32];
    vmem[tid] += vmem[tid + 16];
    vmem[tid] += vmem[tid + 8];
    vmem[tid] += vmem[tid + 4];
    vmem[tid] += vmem[tid + 2];
    vmem[tid] += vmem[tid + 1];
  }
  if (tid == 0) {b[blockIdx.x] = idata[0];}
}
int main(int argc,char*argv[]) {
  int SIZE = 512;
  int N = 1<<20;
  dim3 block(SIZE,1);
  dim3 grid((block.x + N - 1)/block.x, 1);
  int a[N];
  auto init = [&](auto* a,unsigned int size) -> void{
    for(int i = 0;i < size;i++) {
      a[i] = 1;
    }
  };
  int *a_dev, *ans_dev,ans[grid.x];
  init(a, N);
  hipMalloc((int**)(&a_dev),sizeof(int)*N);
  hipMalloc((int**)(&ans_dev),sizeof(int)*grid.x);
  hipMemcpy(a_dev, a, sizeof(int)*N, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  switch(SIZE){
    case 512:
      clock_t start,end;
      start = clock();
      add<512><<<grid.x/8, block>>>(a_dev, ans_dev, N);
      hipDeviceSynchronize();
      end = clock();
      cout<<"GPU time : "<<end - start<<"ms"<<endl;
      break;
  }
  clock_t start,end;
  start = clock();
  for(int i = 1;i < N;i++) {
    a[0] += a[i];
  }
  cout<<a[0]<<endl;
  end = clock();
  cout<<"CPU time : "<<end - start<<"ms"<<endl;
  hipMemcpy (&ans, ans_dev, sizeof(int)*grid.x, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int ret = 0;
  for (int i = 0;i < grid.x;i++) {
    ret += ans[i];
  }
  cout<<ret<<endl;
  return 0;
} 
