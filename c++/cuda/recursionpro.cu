#include "hip/hip_runtime.h"
#include<iostream>
#include<ctime>
//recursion version  of sum
//before: every block's thread0 will generate a grid which contains half threads of father block. time consume :0.014331s
//after:first grid compute the first layer and second grid which has half threads of father grid compute the second layer... 
using namespace std;
template<typename T>
__global__ void add(T* a,T* b,int stride, const int iDim) {
  //iDim is constant val and equal first layer's block.x
  unsigned int tid = threadIdx.x;
  T* idata = a + blockIdx.x * iDim;
  if(stride == 1 && tid == 0) {
    b[blockIdx.x] = idata[0] + idata[1];
    return;
  }
  idata[tid] += idata[tid + stride];
  if(tid == 0 && blockIdx.x == 0) {
    add<<<gridDim.x, stride/2>>>(a, b, stride/2, iDim);
  }
}
int main() {
  int N = 1<<18;
  //int N = 1<<20;
  //problem: when N <= 18 result is valid but else will get a invalid result;
  int SIZE = 1024;
  dim3 block(SIZE,1);
  dim3 grid((block.x + N -1) / block.x,1);
  float a[N],b[grid.x];
  auto init = [&](auto a,int size)->void{
    for(int i = 0;i < size;i++) {
      a[i] = 1;
    }
  };
  init(a, N);
  float *a_dev, *b_dev;
  hipMalloc((float**)&a_dev, sizeof(float)*N);
  hipMalloc((float**)&b_dev, sizeof(float)*grid.x);
  hipMemcpy(a_dev, a, sizeof(float)*N,hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  clock_t start, end;
  start = clock();
  add<<<grid, block.x/2>>>(a_dev, b_dev, block.x/2, block.x);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(b, b_dev, sizeof(float)*grid.x,hipMemcpyDeviceToHost);
  cout<<"GPU use time :"<<end - start<<"ms"<<endl;
  float ans = 0;
  for(int i = 0;i < grid.x;i++) {
    ans += b[i];
  }
  cout<<ans<<endl;
  return 0;
}
