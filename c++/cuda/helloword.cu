
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
#define blockx 8
#define gridx 1
__global__ void print_hello_word(int iSize, int iDepth) {
  int tid = threadIdx.x;
  printf("Recursion=%d: HelloWorld from thread %d \
      block %d\n",iDepth, tid,blockIdx.x);
  if(iSize == 1) {return;}

  int nthreads = iSize>>1;

  if(tid == 0 && nthreads > 0) {
    print_hello_word<<<1,nthreads>>>(nthreads, ++iDepth);
    printf("------> nested execution depth: %d\n",iDepth);
  }
}
int main() {
  dim3 block(blockx, 1);
  dim3 grid(gridx, 1);
  print_hello_word<<<grid, block>>>(8,0);
  hipDeviceSynchronize();
  return 0;
}
