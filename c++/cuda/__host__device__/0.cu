
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int n = 100;
__host__ __device__ bool read(int n) {
  return n != 0;
}

__host__ __device__ bool read0(int n) {
  return n == 0;
}

__global__ void test(int n) {
  if(read0(n)) {
    printf("true\n");
  }
  else if(read(n)){
    printf("false\n");
  }
}
int main() {
  dim3 block(8,2);
        test<<<1,block>>>(n);
        hipDeviceSynchronize();
}
