
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
#define O3
__global__ void add(int*a,int*b,unsigned int n) {
  unsigned int tid = threadIdx.x;
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;
  int *idata = a + blockIdx.x * blockDim.x*8;
  if(idx + 7*blockDim.x < n) {
    a[idx] += a[idx + blockDim.x];
    a[idx] += a[idx + 2*blockDim.x];
    a[idx] += a[idx + 3*blockDim.x];
    a[idx] += a[idx + 4*blockDim.x];
    a[idx] += a[idx + 5*blockDim.x];
    a[idx] += a[idx + 6*blockDim.x];
    a[idx] += a[idx + 7*blockDim.x];
  }
  //if(idx  + blockDim.x < n) a[idx] += a[idx + blockDim.x];
  __syncthreads();
  for(int stride = blockDim.x/2;stride > 32;stride>>=1) {
    if(tid < stride) {
      idata[tid] += idata[tid + stride];
    }
    __syncthreads();
  }
  if(tid < 32) {
    volatile int *vmem = idata;
    vmem[tid] += vmem[tid + 32];
    vmem[tid] += vmem[tid + 16];
    vmem[tid] += vmem[tid + 8];
    vmem[tid] += vmem[tid + 4];
    vmem[tid] += vmem[tid + 2];
    vmem[tid] += vmem[tid + 1];
  }
  if (tid == 0) {b[blockIdx.x] = idata[0];}
}
int main(int argc,char*argv[]) {
  int SIZE =  512;
  int N = 1<<20;
  dim3 block(SIZE,1);
  dim3 grid((block.x + N - 1)/block.x, 1);
  int a[N];
  auto init = [&](auto* a,unsigned int size) -> void{
    for(int i = 0;i < size;i++) {
      a[i] = 1;
    }
  };
  int *a_dev, *ans_dev,ans[grid.x];
  init(a, N);
  hipMalloc((int**)(&a_dev),sizeof(int)*N);
  hipMalloc((int**)(&ans_dev),sizeof(int)*grid.x);
  hipMemcpy(a_dev, a, sizeof(int)*N, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  clock_t start,end;
  start = clock();
  add<<<grid.x/8, block>>>(a_dev, ans_dev, N);
  hipDeviceSynchronize();
  end = clock();
  cout<<"GPU time : "<<end - start<<"ms"<<endl;
  start = clock();
  for(int i = 1;i < N;i++) {
    a[0] += a[i];
  }
  cout<<a[0]<<endl;
  end = clock();
  cout<<"CPU time : "<<end - start<<"ms"<<endl;
  hipMemcpy (&ans, ans_dev, sizeof(int)*grid.x, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int ret = 0;
  for (int i = 0;i < grid.x;i++) {
    ret += ans[i];
  }
  cout<<ret<<endl;
  return 0;
} 
