
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
#define SIZE 64
__global__ void choice(float *array) {
  float a = 0.0,b = 0.0;
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if(index%2) {
    a = 100.0f;
  }else {
    b = 200.0f;
  }
  array[index] = a + b;
  printf("val_choice:%f\n",array[index]);
}
__global__ void choice1(float *array) {
  float a = 0.0f,b = 0.0f;
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if((index/warpSize)%2) {
    a = 100.0f;
  }else {
    b = 200.0f;
  }
  array[index] = a + b;
  printf("val_choice1:%f\n",array[index]);
}
__global__ void compare(float*a,float*b,char *c) {
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if(a[index] != b[index]) {
    *c = 'f';
  }
}
int main() {
  dim3 block(SIZE,1);
  dim3 grid((SIZE + block.x - 1)/block.x,1);
  int n = grid.x*block.x;
  float *a;
  float *b;
  char *c;
  char c_host = 't';
  hipMalloc((float**)&a,sizeof(float)*n);
  hipMalloc((float**)&b,sizeof(float)*n);
  hipMalloc((bool**)&c,sizeof(bool));
  hipMemcpy(c, &c_host, sizeof(char), hipMemcpyHostToDevice);
  choice<<<grid, block>>>(a);
  choice1<<<grid, block>>>(b);
  compare<<<grid, block>>>(a,b,c);
  hipMemcpy(&c_host, c, sizeof(char), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  if(c_host == 'f') {
    cout<<"faild"<<endl;
  }
  else {
    cout<<"success"<<endl;
  }
  hipFree(a);
  hipFree(b);
  hipFree(c);
  return 0;
}
