#include "hip/hip_runtime.h"
#include"./config.cuh"
#include<iostream>
using namespace std;
void __global__ test(int* outdata) {
  int idx = threadIdx.x;
  outdata[idx] = __shfl_xor(idx, 2);
}
int main() {
  MODEL_(one_arg) model{32,1};// one wrap
  int *out,ans[32];
  model(test,&out, 32);
  hipMemcpy(ans, out,sizeof(int)*32, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int i = 0;i < 32;++i) {
    cout << ans[i] << " ";
  }
  cout << "\n";
  return 0;
}
