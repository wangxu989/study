
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
int __device__ warp_reduce(int data) {
  data += __shfl_xor(data,16);
  data += __shfl_xor(data,8);
  data += __shfl_xor(data,4);
  data += __shfl_xor(data,2);
  data += __shfl_xor(data,1);
  return data;
}
void __global__ reduce(int* indata, int* outdata, int n, int warps) {
  extern __shared__ int sm[];
  int laneid = threadIdx.x % 32;
  int warpid = threadIdx.x / 32;
  int idx = blockDim.x * blockIdx.x*4 + threadIdx.x;
  int data = indata[idx];
  int data1 = indata[idx + blockDim.x];
  int data2 = indata[idx + blockDim.x*2];
  int data3 = indata[idx + blockDim.x*3];
  data = warp_reduce(data) + warp_reduce(data1) + warp_reduce(data2) + warp_reduce(data3);
  //reduce global
   if(laneid == 0) {
    sm[warpid] = data;
  }
  __syncthreads();
  //reduce shm
  data = threadIdx.x < warps ? sm[threadIdx.x]:0;
  data = warp_reduce(data);
  if(threadIdx.x == 0) {
    atomicAdd(outdata,data);
  }
}
int main() {
  int n = 1 << 22;
  dim3 block(128, 1);
  dim3 grid((n + block.x - 1)/block.x/4, 1);
  int* in_dev, *in;
  int* out_dev, out;
  in = (int*)malloc(n*sizeof(int));
  for(int i = 0;i < n;++i) {
    in[i] = 1;
  }
  clock_t start ,end;
  hipError_t error = hipMalloc((void**)&in_dev, sizeof(int)*n);
  cout << "error : " <<error << endl;
  hipMalloc((void**)&out_dev, sizeof(int));
  hipMemcpy(in_dev, in, sizeof(int)*n, hipMemcpyHostToDevice);
  start = clock();
  reduce<<<grid,block,block.x/32>>>(in_dev, out_dev, n, block.x/32);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(&out, out_dev,sizeof(out), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int ans = 0;
  cout << "gpu time :" << end -start << "\n";
  cout << "ans :" << out << "\n";
  return 0;
}
