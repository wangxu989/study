
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
#define O1
__global__ void add(int *a,int*b,unsigned int n) {
  unsigned int tid = threadIdx.x;
  int *idata = a + blockIdx.x * blockDim.x;
  unsigned int idx = tid + blockIdx.x * blockDim.x ;
  if(idx >= n) {
    //printf("%d,",blockIdx.x);
    return;
  }
  //printf("%d, ",a[idx]);
#ifdef O3
  for(int stride = blockDim.x/2;stride > 0;stride>>=1) {
    if(tid < stride) {
      idata[tid] += idata[tid + stride];
    }
    __syncthreads();
  }
#else
  for (int i = 1;i < blockDim.x;i *= 2) {
#ifdef O1
    if((tid %(2 * i)) == 0) {
      idata[tid] += idata[tid + i];  
    }
#endif
#ifdef O2
    int index = 2*i*tid;
    if(index < blockDim.x) {
      idata[index] += idata[index + i];
    }
#endif
    __syncthreads();

  }
#endif
  if (tid == 0) {b[blockIdx.x] = idata[0];}
}
int main(int argc,char*argv[]) {
  int SIZE = 512;
  int N = 1<<20;
  dim3 block(SIZE,1);
  int num_gridx = (block.x + N - 1) / block.x;
  dim3 grid(num_gridx, 1);
  std::cout<<"grid: "<<grid.x<<" block: "<<block.x<<std::endl;
  int a[N];
  auto init = [&](auto* a,unsigned int size) -> void{
    for(int i = 0;i < size;i++) {
      //a[i] = random()%100;
      a[i] = 1;
    }
  };
  int *a_dev, *ans_dev,ans[grid.x];
  init(a, N);
  hipMalloc((int**)(&a_dev),sizeof(int)*N);
  hipMalloc((int**)(&ans_dev),sizeof(int)*grid.x);
  hipMemcpy(a_dev, a, sizeof(int)*N, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  clock_t start,end;
  start = clock();
  add<<<grid, block>>>(a_dev, ans_dev, N);
  hipDeviceSynchronize();
  end = clock();
  cout<<"GPU time : "<<end - start<<"ms"<<endl;
  start = clock();
  for(int i = 1;i < N;i++) {
    a[0] += a[i];
  }
  cout<<a[0]<<endl;
  end = clock();
  cout<<"CPU time : "<<end - start<<"ms"<<endl;
  hipMemcpy (&ans, ans_dev, sizeof(int)*grid.x, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int ret = 0;
  for (int i = 0;i < grid.x;i++) {
    ret += ans[i];
  }
  cout<<ret;
  return 0;
} 
