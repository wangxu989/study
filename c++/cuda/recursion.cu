#include "hip/hip_runtime.h"
#include<iostream>
#include<ctime>
//recursion version  of sum
using namespace std;
template<typename T>
__global__ void add(T* a,T* b,unsigned int iSize) {
  unsigned int tid = threadIdx.x;
  T* idata = a + blockDim.x * blockIdx.x;
  T* odata = b + blockIdx.x;
  if(iSize == 2 && tid == 0) {
    *b = idata[0] + idata[1];   
    return;
  }
  int stride = iSize>>1;
  if (stride > 1 && tid < stride) {
    idata[tid] += idata[tid + stride];
  }
  //__syncthreads();
  if(tid == 0) {
    //idata & odata
    //why ? 
    //odata : because after depth0 blockIdx.x = 0
    add<<<1, stride>>>(idata, odata, stride);
    //hipDeviceSynchronize();
  }
 // __syncthreads();
}
int main() {
  int N = 1<<20;
  int SIZE = 512;
  dim3 block(SIZE,1);
  dim3 grid((block.x + N -1) / block.x,1);
  float a[N],b[grid.x];
  auto init = [&](auto a,unsigned int size)->void{
    for(int i = 0;i < size;i++) {
      a[i] = 1;
    }
  };
  init(a, N);
  float *a_dev, *b_dev;
  hipMalloc((float**)&a_dev, sizeof(float)*N);
  hipMalloc((float**)&b_dev, sizeof(float)*grid.x);
  hipMemcpy(a_dev, a, sizeof(float)*N,hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  clock_t start, end;
  start = clock();
  add<<<grid, block>>>(a_dev, b_dev, block.x);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(b, b_dev, sizeof(float)*grid.x,hipMemcpyDeviceToHost);
  cout<<"GPU use time :"<<end - start<<"ms"<<endl;
  float ans = 0;
  for(int i = 0;i < grid.x;i++) {
    ans += b[i];
  }
  cout<<ans<<endl;
  return 0;
}
