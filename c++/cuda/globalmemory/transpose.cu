#include "hip/hip_runtime.h"
#include<iostream>
#include<ctime>
#include"head.h"
using namespace std;
#define R
template<typename T>
void __global__ transpose(const T* src,T *after,unsigned int row, unsigned int column) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x * 4;
  unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if(idx + blockDim.x * blockIdx.x * 4< row && idy < column) {
    //read merge
#ifdef R
    unsigned int fo = idx * column + idy;
    unsigned int to = idy * row + idx;
    //read column write row
    after[to] = src[fo];
    after[to + blockDim.x] = src[fo + blockDim.x];
    after[to + blockDim.x*2] = src[fo + blockDim.x*2];
    after[to + blockDim.x*3] = src[fo + blockDim.x*3];
#elif define C
    //read row write column
    after[idx * column + idy] = src[idy * row + idx];
    //after[idx * column + idy] = src[idy * row + idx];

#endif
  }
}
template<typename T>
void transposeHost(const T* src, T* after, unsigned int row, unsigned int column) {
  for (int i = 0;i < row;i++) {
    for(int j = 0;j < column;j++) {
      after[j * row + i] = src[i * column + j];
    }
  }
}
int main(int argc,char *argv[]) {
  int Row = 32;
  int Column = 16;
  int nx = 1 << 9;
  int ny = 1 << 9;
  int a[] = {0,1,2,3,4,5,6,7,8,9,10,11};
  int b[12];
  memset(b,0x0,sizeof(b));
  transposeHost(a,b,3,4);
  for (int i = 0;i < 12;i++) {
    cout << b[i] <<" ";
  }
  cout <<endl;
  int a_[ny * nx],*a_dev,*b_dev;
  dim3 block(Row,Column);
  dim3 grid( (nx + Row*4 - 1)/Row*4,  (Column + ny - 1)/Column);
  hipMalloc((void**)&a_dev,sizeof(a_));
  hipMalloc((void**)&b_dev,sizeof(a_));
  cout << "brea 1" << sizeof(a_) <<endl;
  hipMemcpy(a_dev,a_,sizeof(a_),hipMemcpyHostToDevice);
  clock_t start, end;
  start = clock();
  transpose<<<block,grid>>>(a_dev,b_dev,nx,ny);
  hipDeviceSynchronize();
  end = clock();
  cout << end - start << "ms" << endl;
  return 0;
}
