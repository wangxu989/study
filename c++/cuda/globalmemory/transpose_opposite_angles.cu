
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
using namespace std;
typedef unsigned int index_t;
void __global__ transpose(float *src, float *target, index_t nx, index_t ny) {
  index_t blk_y = blockIdx.x;
  index_t blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

  index_t ix = blockDim.x * blk_x + threadIdx.x;
  index_t iy = blockDim.y * blk_y + threadIdx.y;

  if(ix < nx && iy < ny) {
    target[ix * ny + iy] = src[iy * nx + ix];
  }
}
void __device__ __host__ transposeHost(float *src, float *target, index_t row ,index_t column) {
  for (index_t i = 0;i < row;++i) {
    for (index_t j = 0;j < column;++j) {
      target[j * column + i] = src[i * row + j];
    }
  }
}
int main() {
  int N = 1 << 18;
  cout << N;
  int nx = 1 << 9;
  int ny = 1 << 9;
  int blockx = 32;
  int blocky = 32;
  clock_t start, end;
  float a[N],b[N];
  for(int i = 0;i < N;i++) {
    a[i] = i;
  }
  start = clock();
  transposeHost(a, b, nx,ny);
  end = clock();
  cout << "cpu time:" << end -start <<endl;
  float *a_dev, *b_dev;
  hipMalloc((void**)&a_dev, sizeof(a));
  hipMalloc((void**)&b_dev, sizeof(a));
  hipMemcpy(a_dev, a, sizeof(a), hipMemcpyHostToDevice);
  dim3 block(blockx, blocky);
  dim3 grid((nx + blockx - 1)/blockx,(ny + blocky - 1)/blocky);
  start = clock();
  transpose<<<block, grid>>>(a_dev, b_dev, nx, ny);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(a,b_dev, sizeof(a), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  cout << "gpu time:" << end -start <<endl;
  for(int i = 0;i < N;i++) {
    if(a[i] != b[i]) {
      cout << a[i] << "   "<<b[i] <<endl;
    }
  }
  return 0;
}
