
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
#define Size 512
using namespace std;
template<typename T, unsigned int BlockSize>
void __global__ add(const T* lhs,const T *rhs ,T*sum, const unsigned int n) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x*4;
  if(idx + 3*blockDim.x < n) {
    sum[idx] = lhs[idx] + rhs[idx];
    sum[idx + blockDim.x] = lhs[idx + blockDim.x] + rhs[idx + blockDim.x];
    sum[idx + blockDim.x*2] = lhs[idx + blockDim.x*2] + rhs[idx + blockDim.x*2];
    sum[idx + blockDim.x*3] = lhs[idx + blockDim.x*3] + rhs[idx + blockDim.x*3];
  }
}
int main() {
  int N = 1 << 19;
  dim3 block(Size,1);
  dim3 grid( (N + Size - 1)/Size/4 ,1);
  float  a[N],b[N],c[N] ,*a_dev,*b_dev,*c_dev;
  clock_t start, end;
  memset(c, 0 , sizeof(c));
  auto init = [&](auto* a,unsigned int n) {
    for (int i = 0;i < n;i++) {
      a[i] = (float)(rand()&0xff) / 100.0f;
    }
  };
  init(a,N);
  init(b,N);
  hipMalloc((void**)&a_dev, sizeof(float)*N);
  hipMalloc((void**)&b_dev, sizeof(float)*N);
  hipMalloc((void**)&c_dev, sizeof(float)*N);
  hipMemcpy(a_dev ,a , sizeof(a),hipMemcpyHostToDevice);
  hipMemcpy(b_dev ,b , sizeof(b),hipMemcpyHostToDevice);
  start = clock();
  add<float,Size><<<grid, block>>>(a_dev, b_dev, c_dev,N);
  hipDeviceSynchronize();
  end = clock();
  cout << "sum time on gpu:" << end - start << endl;

  hipMemcpy(c, c_dev,sizeof(c),hipMemcpyDeviceToHost);
  for (int i = 0;i < N;i++) {
    if(a[i] + b[i] != c[i]) {
      cout << "failed" <<endl;
    }
  }
  return 0;
}
