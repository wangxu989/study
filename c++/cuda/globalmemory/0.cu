#include "hip/hip_runtime.h"
#include<iostream>
#include"./include/check_input.h"
#include"./include/prints.h"
using namespace std;
using namespace check;
using namespace prints;
__device__ float val;
void __global__ test() {
  printf("test_global_val: %f\n",val);
}
int main(int argc ,char* argv[]) {
  device_conf dev{1,1};
  check_input(argc, argv, dev);
  print_v(dev.blocksize, dev.gridsize);
  float t = 2.0f;
  hipMemcpyToSymbol(HIP_SYMBOL(val), &t, sizeof(float));
  dim3 block(dev.blocksize, 1);
  dim3 grid(dev.gridsize, 1);
  test<<<grid, block>>>();
  hipDeviceSynchronize();
}

