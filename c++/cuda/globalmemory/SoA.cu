#include "hip/hip_runtime.h"
#include<iostream>
#include<ctime>
using namespace std;
#define Size 512
struct SoA {
  float x[N];
  float y[N];
};
void __global__  test_SoA(struct SoA *data,unsigned int n) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    data->x[idx] += 1.0f;
    data->y[idx] += 2.0f;
  }
}
int main() {
  int N = 1 << 18;
  dim3 block(Size, 1);
  dim3 grid((Size + N - 1)/Size, 1);
  
  struct SoA a , *a_dev;
  hipMalloc((void**)(&a_dev),sizeof(struct SoA));
  hipMemcpy(a_dev, &a ,sizeof(struct SoA), hipMemcpyHostToDevice);
  clock_t start ,end;
  start = clock();
  test_SoA<<<block,grid>>>(a_dev,N);
  hipDeviceSynchronize();
  end = clock();
  cout << "sum time gpu compute:" << end -start << endl;
  hipDeviceReset();
  return 0;
}
