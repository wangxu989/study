
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
#define exp 20
#define Size 512
using namespace std;
struct AoS{
  float x,y;
};
void __global__ AoS(AoS* data,unsigned int n) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    data[idx].x += 1.0f;
    data[idx].y += 2.0f;
  }
}
int main() {
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  cout << "device " << dev << ": " << deviceProp.name << endl;
  int N = 1 << 18; 
  dim3 block(Size,1);
  dim3 grid((Size + N - 1) / Size, 1);
  struct AoS a[N],*a_dev;
  hipMalloc((void**)&a_dev, sizeof(struct AoS)*N);
  hipMemcpy(a_dev, a, sizeof(a), hipMemcpyHostToDevice);
  clock_t start ,end;
  start = clock();
  AoS<<<grid, block>>>(a_dev, N);
  hipDeviceSynchronize();
  end = clock();
  cout << "sum time in gpu compute:" << end - start << "ms" << endl;
  hipFree(a_dev);
  hipDeviceReset();
  return 0;
}
