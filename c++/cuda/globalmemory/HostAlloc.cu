#include "hip/hip_runtime.h"
#include<iostream>
#include"./head.h"
using namespace std;
template<typename T, unsigned int blocksize>
void __global__ add(T* data,T* odata, unsigned int n) {
  unsigned int tid = threadIdx.x;
  T* idata = data + blockDim.x * blockIdx.x * 8;
  unsigned id = threadIdx.x + blockDim.x *blockIdx.x * 8;
  //展开
  if (id + 7*blockDim.x < n) {
    idata[tid] += idata[tid + blockDim.x];
    idata[tid] += idata[tid + 2*blockDim.x];
    idata[tid] += idata[tid + 3*blockDim.x];
    idata[tid] += idata[tid + 4*blockDim.x];
    idata[tid] += idata[tid + 5*blockDim.x];
    idata[tid] += idata[tid + 6*blockDim.x];
    idata[tid] += idata[tid + 7*blockDim.x];
  }
  __syncthreads();
  if(blocksize >= 512 && tid < 256) {
    idata[tid] += idata[tid + 256];
  }
  __syncthreads();
  if(blocksize >= 256 && tid < 128) {
    idata[tid] += idata[tid + 128];
  }
  __syncthreads();
  if(blocksize >= 128 && tid < 64) {
    idata[tid] += idata[tid + 64];
  }
  __syncthreads();
  if(tid < 32) {
    volatile T * vmem = idata;
    vmem[tid] += vmem[tid + 32];
    vmem[tid] += vmem[tid + 16];
    vmem[tid] += vmem[tid + 8];
    vmem[tid] += vmem[tid + 4];
    vmem[tid] += vmem[tid + 2];
    vmem[tid] += vmem[tid + 1];
  }
  if (tid == 0) {
    odata[blockIdx.x] = idata[0];
  }
}
int main() {
  int N = 1<<20;
  int SIZE = 512;
  dim3 block(SIZE , 1);
  dim3 grid((N + block.x - 1)/block.x , 1);
  float* h_a, *h_b;
  float *d_a, *d_b;
  hipHostAlloc((void **)&h_a, N*sizeof(float), hipHostMallocMapped);
  hipHostAlloc((void **)&h_b, grid.x/8, hipHostMallocMapped);
  auto init = [&](auto a,unsigned int n)->void{
    for(int i = 0;i < n;i++) {
      a[i] = 1;
    }
  };
  init(h_a, N);
  init(h_b, grid.x/8);
  hipHostGetDevicePointer((void**)&d_a, h_a, 0);
  hipHostGetDevicePointer((void**)&d_b, h_b, 0);
  clock_t start, end;
  start = clock();
  add<float,512><<<grid.x/8, block>>>(d_a, d_b, N);
  hipDeviceSynchronize();
  end = clock();
  cout << "GPU Time is :" << end - start << '\n';
  float ans = 0.0f;
  for(int i = 0;i < grid.x/8;i++) {
    ans += h_b[i];
  }
  cout << ans << '\n';
  return 0;
}
