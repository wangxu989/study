#include "hip/hip_runtime.h"
#include<iostream>
#include<ctime>
#include"head.h"
using namespace std;
#define R 
template<typename T>
void __global__ transpose(const T* src,T *after,unsigned int row, unsigned int column) {
  unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x * 4;
  unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if(idx + blockDim.x  * 3< row && idy < column) {
    //read merge
#ifdef C 
    unsigned int to = idx * column + idy;
    unsigned int fo = idy * row + idx;
    unsigned int step = blockDim.x * column;
    //read row write column
    after[to] = src[fo];
    after[to + step] = src[fo + blockDim.x];
    after[to + step*2] = src[fo + blockDim.x*2];
    after[to + step*3] = src[fo + blockDim.x*3];
#else 
    //read column write row
    unsigned int to = idx * column + idy;
    unsigned int fo = idy * row + idx;
    unsigned int step = blockDim.x * column;
    after[fo] = src[to];
    after[fo + blockDim.x] = src[to + step];
    after[fo + blockDim.x*2] = src[to + step*2];
    after[fo + blockDim.x*3] = src[to + step*3];
#endif
  }
}
template<typename T>
void transposeHost(const T* src, T* after, unsigned int row, unsigned int column) {
  for (int i = 0;i < row;i++) {
    for(int j = 0;j < column;j++) {
      after[j * row + i] = src[i * column + j];
    }
  }
}
int main(int argc,char *argv[]) {
  int Row = 32;
  int Column = 32;
  int nx = 1 << 9;
  int ny = 1 << 9;
  int N = nx * ny;
  int a[N], b[N],*a_dev,*b_dev;
  clock_t start, end;
  for(int i = 0;i < N;i++) {
    a[i] = i;
  }
  start = clock();
  transposeHost(a,b,nx,ny);
  end = clock();
  cout << "cpu :" << end - start << "ms" << endl;
  dim3 block(Row,Column);
  dim3 grid( (nx + Row*4 - 1)/Row/4,  (Column + ny - 1)/Column);
  hipMalloc((void**)&a_dev,sizeof(a));
  hipMalloc((void**)&b_dev,sizeof(a));
  cout << "brea 1" << sizeof(a) <<endl;
  hipMemcpy(a_dev,a,sizeof(a),hipMemcpyHostToDevice);
  start = clock();
  transpose<<<block,grid>>>(a_dev,b_dev,nx,ny);
  hipDeviceSynchronize();
  end = clock();
  hipMemcpy(a, b_dev, sizeof(a), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for(int i = 0;i < N;i++) {
    if(a[i] != b[i]) {
      cout <<"failed" << endl;
    }
  }
  cout <<"gpu:" <<  end - start << "ms" << endl;
  return 0;
}
